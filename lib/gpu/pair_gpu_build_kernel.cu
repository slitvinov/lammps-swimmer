#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Peng Wang (Nvidia), penwang@nvidia.com
                         Mike Brown (ORNL), brownw@ornl.gov
------------------------------------------------------------------------- */

#ifdef NV_KERNEL

#include "geryon/ucl_nv_kernel.h"
texture<float4> neigh_tex;

#ifdef _DOUBLE_DOUBLE
__inline double4 fetch_pos(const int i, const double4 *pos)
{
  return pos[i];
}
#else
__inline float4 fetch_pos(const int& i, const float4 *pos)
{
  return tex1Dfetch(neigh_tex, i);
}
#endif

#else

#define fetch_pos(i,y) x_[i]

#endif

#ifdef _DOUBLE_DOUBLE
#define numtyp double
#define numtyp4 double4
#endif

#ifdef _SINGLE_DOUBLE
#define numtyp float
#define numtyp4 float4
#endif

#ifndef numtyp
#define numtyp float
#define numtyp4 float4
#endif

#define CELL_BLOCK_SIZE 64
#define BLOCK_2D 8

__kernel void transpose(int *out, int *in, int columns_in, int rows_in)
{
	__local float block[BLOCK_2D][BLOCK_2D+1];
	
	unsigned ti=THREAD_ID_X;
	unsigned tj=THREAD_ID_Y;
	unsigned bi=BLOCK_ID_X;
	unsigned bj=BLOCK_ID_Y;
	
	unsigned i=bi*BLOCK_2D+ti;
	unsigned j=bj*BLOCK_2D+tj;
	if ((i<columns_in) && (j<rows_in))
		block[tj][ti]=in[j*columns_in+i];

	__syncthreads();

	i=bj*BLOCK_2D+ti;
	j=bi*BLOCK_2D+tj;
	if ((i<rows_in) && (j<columns_in))
		out[j*rows_in+i] = block[ti][tj];
}

__kernel void calc_cell_id(numtyp4 *pos, unsigned *cell_id, int *particle_id,
                           numtyp boxlo0, 
                           numtyp boxlo1, numtyp boxlo2, numtyp boxhi0, 
                           numtyp boxhi1, numtyp boxhi2, numtyp cell_size, 
                           int ncellx, int ncelly, int nall) {
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i < nall) {
    numtyp4 p = fetch_pos(i,pos); //pos[i];

    p.x -= boxlo0;
    p.y -= boxlo1;
    p.z -= boxlo2;
    
    p.x = fmaxf(p.x, -cell_size);
    p.x = fminf(p.x, boxhi0-boxlo0+cell_size);
    p.y = fmaxf(p.y, -cell_size);
    p.y = fminf(p.y, boxhi1-boxlo1+cell_size);
    p.z = fmaxf(p.z, -cell_size);
    p.z = fminf(p.z, boxhi2-boxlo2+cell_size);
    
    unsigned int id = (unsigned int)(p.x/cell_size + 1.0) 
      + (unsigned int)(p.y/cell_size + 1.0) * ncellx
      + (unsigned int)(p.z/cell_size + 1.0) * ncellx * ncelly;
    
    cell_id[i] = id;
    particle_id[i] = i;
  }
}

__kernel void kernel_calc_cell_counts(unsigned *cell_id,
                                      int *cell_counts, int nall, int ncell) {
  int idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (idx < nall) {
    int id = cell_id[idx];

    // handle boundary cases
    if (idx == 0) {
      for (int i = 0; i < id + 1; i++) 
        cell_counts[i] = 0;
    }
    if (idx == nall - 1) {
      for (int i = id+1; i <= ncell; i++) 
        cell_counts[i] = nall;
    }

    if (idx > 0 && idx < nall) {
      int id_l = cell_id[idx-1];
      if (id != id_l) {
        for (int i = id_l+1; i <= id; i++) 
          cell_counts[i] = idx;
      }
    }
  }
}

__kernel void calc_neigh_list_cell(numtyp4 *pos,
				     int *cell_particle_id, 
				     int *cell_counts,
				     int *nbor_list,
				     int *host_nbor_list, 
				     int neigh_bin_size, 
				     numtyp cell_size,
				     int ncellx, int ncelly, int ncellz,
				     int inum, int nt, int nall)
{
  int tid = threadIdx.x;
  int ix = blockIdx.x;
  int iy = blockIdx.y % ncelly;
  int iz = blockIdx.y / ncelly;
	  
  int icell = ix + iy*ncellx + iz*ncellx*ncelly;

  __shared__ int cell_list_sh[CELL_BLOCK_SIZE];
  __shared__ numtyp4 pos_sh[CELL_BLOCK_SIZE];

  int icell_begin = cell_counts[icell];
  int icell_end = cell_counts[icell+1];

  int nborz0 = max(iz-1,0), nborz1 = min(iz+1, ncellz-1),
      nbory0 = max(iy-1,0), nbory1 = min(iy+1, ncelly-1),
      nborx0 = max(ix-1,0), nborx1 = min(ix+1, ncellx-1);

  numtyp4 diff;
  numtyp r2;
  for (int ii = 0; ii < ceil((numtyp)(icell_end - icell_begin)/blockDim.x); ii++) {
    int i = icell_begin + tid + ii*blockDim.x;
    int pid_i = nall, pid_j, stride;
    numtyp4 atom_i, atom_j;
    int cnt = 0;    
    int *neigh_counts, *neigh_list;
    
    if (i < icell_end)
      pid_i = cell_particle_id[i];

    if (pid_i < nt) {
      atom_i = fetch_pos(pid_i,pos); //pos[pid_i];
    }
    if (pid_i < inum) {
      stride=inum;
      neigh_counts=nbor_list+stride+pid_i;
      neigh_list=neigh_counts+stride;
      nbor_list[pid_i]=pid_i;
    } else {
      stride=nt-inum;
    	neigh_counts=host_nbor_list+pid_i-inum;
      neigh_list=neigh_counts+stride;
    }
    
    // loop through neighbors

    for (int nborz = nborz0; nborz <= nborz1; nborz++) {
      for (int nbory = nbory0; nbory <= nbory1; nbory++) {
        for (int nborx = nborx0; nborx <= nborx1; nborx++) {
	
          int jcell = nborx + nbory*ncellx + nborz*ncellx*ncelly;
		
          int jcell_begin = cell_counts[jcell];
          int jcell_end = cell_counts[jcell+1];
          int num_atom_cell = jcell_end - jcell_begin;
	  
          // load jcell to shared memory
          int num_iter = (int)ceil((numtyp)num_atom_cell/CELL_BLOCK_SIZE);

          for (int k = 0; k < num_iter; k++) {
            int end_idx = min(CELL_BLOCK_SIZE, num_atom_cell-k*CELL_BLOCK_SIZE);
	    
            if (tid < end_idx) {
              pid_j =  cell_particle_id[tid+k*CELL_BLOCK_SIZE+jcell_begin];
              cell_list_sh[tid] = pid_j;
              atom_j = fetch_pos(pid_j,pos); //[pid_j];
              pos_sh[tid].x = atom_j.x;
              pos_sh[tid].y = atom_j.y;
              pos_sh[tid].z = atom_j.z;
            }
            __syncthreads();
	    
            if (pid_i < nt) {
	    
              for (int j = 0; j < end_idx; j++) {
                int pid_j = cell_list_sh[j]; // gather from shared memory
                if (pid_i<inum || pid_j<inum || pid_j>pid_i) {
                  diff.x = atom_i.x - pos_sh[j].x;
                  diff.y = atom_i.y - pos_sh[j].y;
                  diff.z = atom_i.z - pos_sh[j].z;
		
                  r2 = diff.x*diff.x + diff.y*diff.y + diff.z*diff.z;
                  if (r2 < cell_size*cell_size && r2 > 1e-5) {
                    if (cnt < neigh_bin_size) {
                      *neigh_list = pid_j;
                      neigh_list+=stride;
                    }
                    cnt++;
                  }		
                }
              }
            }
	          __syncthreads();
	        } // for (k)
        }
      }
    }
    if (pid_i < nt)
      *neigh_counts = cnt;
  } // for (i)
}

__kernel void kernel_special(__global int *dev_nbor, 
                             __global int *host_nbor_list, __global int *tag,
                             __global int *nspecial, __global int *special,
                             int inum, int nt, int nall) {
  // ii indexes the two interacting particles in gi
  int ii=GLOBAL_ID_X;

  if (ii<nt) {
    int stride;
    __global int *list, *list_end;
    
    int n1=nspecial[ii*3];
    int n2=nspecial[ii*3+1];
    int n3=nspecial[ii*3+2];

    if (ii < inum) {
      stride=inum;
      list=dev_nbor+stride+ii;
    } else {
      stride=nt-inum;
      list=host_nbor_list+ii-inum;
    }
    int numj=*list;
    list+=stride;
    list_end=list+numj*stride;
  
    for ( ; list<list_end; list+=stride) {
      int nbor=*list;
      int jtag=tag[nbor];

      int offset=ii;
      for (int i=0; i<n3; i++) {
        if (special[offset]==jtag) {
          nbor+=nall;
          if (i>=n1)
            nbor+=nall;
          if (i>=n2)
            nbor+=nall;
        }
        offset+=nt;
      }
      if (nbor>=nall)
        *list=nbor;
    }
  } // if ii
}

