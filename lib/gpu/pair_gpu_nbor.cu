/***************************************************************************
                               pair_gpu_nbor.cu
                             -------------------
                               W. Michael Brown

  Neighbor memory operations for LAMMPS GPU Library

 __________________________________________________________________________
    This file is part of the LAMMPS GPU Library
 __________________________________________________________________________

    begin                : Tue Aug 4 2009
    copyright            : (C) 2009 by W. Michael Brown
    email                : wmbrown@sandia.gov
 ***************************************************************************/

/* -----------------------------------------------------------------------
   Copyright (2009) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.
   ----------------------------------------------------------------------- */

#include "pair_gpu_nbor.h"

int PairGPUNbor::bytes_per_atom(const int max_nbors) const {
  if (_use_packing)
    return (max_nbors*2+4)*sizeof(int);
  else
    return (max_nbors+3)*sizeof(int);
}

void PairGPUNbor::init(const int ij_size, const int max_atoms, 
                       const int max_nbors) {
  if (allocated)
    clear();
    
  // Initialize timers for the selected GPU
  time_nbor.init();

  if (_use_packing)
    dev_nbor.safe_alloc(max_nbors+4,max_atoms);
  else  
    dev_nbor.safe_alloc(3,max_atoms);
  
  ij.safe_alloc(max_nbors*max_atoms);
  host_ij.safe_alloc_w(ij_size);
    
  allocated=true;
}
  
void PairGPUNbor::clear() {
  if (!allocated)
    return;
  allocated=false;

  ij.clear();
  host_ij.clear();
  dev_nbor.clear();
}  
 
double PairGPUNbor::host_memory_usage() const {
  return IJ_SIZE*sizeof(int)+sizeof(PairGPUNbor);
}

void PairGPUNbor::reset(const int inum, int *ilist, const int *numj, 
                        hipStream_t &s) {  
  ij_total=0;

  dev_nbor.copy_from_host(ilist,inum);
  int acc=0;
  for (int i=0; i<inum; i++) {
    host_ij[i]=numj[ilist[i]];
    host_ij[i+inum]=acc;
    acc+=numj[ilist[i]];
  }
  
  host_ij.copy_to_2Ddevice(dev_nbor.begin()+dev_nbor.row_size(),
                           dev_nbor.row_size(),2,inum, s);
}
