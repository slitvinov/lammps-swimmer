#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (SNL), wmbrown@sandia.gov
                         Peng Wang (Nvidia), penwang@nvidia.com
                         Paul Crozier (SNL), pscrozi@sandia.gov
------------------------------------------------------------------------- */
#include <iostream>
#include <cassert>
#include <string.h>
#include "cudatimer.h"
#include "lj_tex.h"
#include "neigh.h"
#include "cell.h"
#include "lj_gpu_kernel.h"

#ifdef WINDLL
#define EXTERN extern "C" __declspec(dllexport) 
#else
#define EXTERN 
#endif

static float h_boxlo[3], h_boxhi[3];
static float cell_size;
static float *energy = NULL, *d_energy = NULL;
static float3 *d_force = NULL, *f_temp = NULL, *v_temp = NULL, *d_virial = NULL;
static float4 *d_pos = NULL, *temp_pos = NULL;
static int *d_type = NULL;
static int ncellx, ncelly, ncellz;

static neigh_list_gpu d_neigh_list;
static cell_list_gpu d_cell_list;

#define TIMING(x) 

// ---------------------------------------------------------------------------
// Return string with GPU info
// ---------------------------------------------------------------------------
EXTERN void lj_gpu_name(const int id, const int max_nbors, char * name) 
{
  struct hipDeviceProp_t prop;
  CUDA_SAFE_CALL( hipGetDeviceProperties(&prop, id) );
#ifdef _WIN32
  strcpy_s(name, strlen(prop.name)+1, prop.name);
#else
  strncpy(name, prop.name, strlen(prop.name)+1);
#endif
}

// ---------------------------------------------------------------------------
// Allocate memory on host and device and copy constants to device
// ---------------------------------------------------------------------------
EXTERN bool lj_gpu_init(int &ij_size, const int ntypes, 
			double **cutsq,double **sigma, 
			 double **epsilon, double **host_lj1, double **host_lj2, 
			 double **host_lj3, double **host_lj4, double **offset, 
			 double *special_lj, double *boxlo, double *boxhi, 
			 double cellsize, double skin,
			 const int max_nbors, const int gpu_id) 
{
  int num_devices;

  /* get device count */
  CUDA_SAFE_CALL( hipGetDeviceCount(&num_devices) );
  if (num_devices == 0) {
    printf("NO CUDA-capable GPU detected.\n");
    exit(1);
  }

  if (gpu_id > num_devices) {
    printf("gpu_id %d is larger than the number of GPUs %d\n", 
	   gpu_id, num_devices);
    exit(1);
  }

  /* set CUDA device to the specified GPU */
  hipDeviceReset();
  CUDA_SAFE_CALL( hipSetDevice(gpu_id) );
  
  ij_size=0;

  cell_size = cellsize;
  ncellx = ceil(((boxhi[0] - boxlo[0]) + 2.0*cell_size) / cell_size);
  ncelly = ceil(((boxhi[1] - boxlo[1]) + 2.0*cell_size) / cell_size);
  ncellz = ceil(((boxhi[2] - boxlo[2]) + 2.0*cell_size) / cell_size);
   
  for (int i = 0; i < 3; i++) {
    h_boxhi[i] = boxhi[i];
    h_boxlo[i] = boxlo[i];
  }

  init_force_const(ntypes, cutsq, host_lj1, host_lj2, host_lj3, host_lj4, offset);

  init_cell_list_const(cellsize, skin, boxlo, boxhi);

  return true;
}

// ---------------------------------------------------------------------------
// Clear memory on host and device
// ---------------------------------------------------------------------------
EXTERN void lj_gpu_clear() {

  free(energy);
  free(v_temp);
  CUDA_SAFE_CALL( hipHostFree(f_temp) );
  if (d_force) CUDA_SAFE_CALL( hipFree(d_force) );
  if (d_energy) CUDA_SAFE_CALL( hipFree(d_energy) );
  if (d_virial) CUDA_SAFE_CALL( hipFree(d_virial) );
  if (d_pos) CUDA_SAFE_CALL( hipFree(d_pos) );
  if (d_type) CUDA_SAFE_CALL( hipFree(d_type) );
  if (temp_pos) CUDA_SAFE_CALL( hipHostFree(temp_pos) );
  clear_neigh_list_gpu(d_neigh_list);
  clear_cell_list_gpu(d_cell_list);

  if (useCache) {
    unbind_pos();
    unbind_type();
  }


  //LJMF.clear();
}


template <class numtyp, class acctyp>
double _lj_gpu_neigh(double **force, double *virial,
		     double **host_x, int *host_type, const int inum, 
		     const int nall, const int ago, const bool eflag, const bool vflag, 
		     const double *boxlo, const double *boxhi)
{

  double evdwl=0.0;

  static int first_call = 1;
  
  TIMING( static CUDATimer cuTimer );  
  TIMING( static CTimer cTimer );
  TIMING( static CTimer cTimer2 );
  
  double *atom_pos = host_x[0];

  static int szTailList = inum*32;
  
  TIMING( cTimer.Start() );
  TIMING( cTimer2.Start() );
   
  /* MPI communication just happened, reallocate space using new inum & nall
     FIXME: this is costly: ~ total kernel time! Use a DIY GPU memory allocator.*/

  if (first_call || ago == 0) {

    if (!first_call) {
      if (useCache) {
	unbind_pos();
	unbind_type();
      }
      
      CUDA_SAFE_CALL( hipFree(d_force) );
      CUDA_SAFE_CALL( hipFree(d_energy) );
      CUDA_SAFE_CALL( hipFree(d_virial) );
      CUDA_SAFE_CALL( hipFree(d_pos) );
      CUDA_SAFE_CALL( hipFree(d_type) );

      clear_neigh_list_gpu(d_neigh_list);

      CUDA_SAFE_CALL( hipHostFree(f_temp) );
      CUDA_SAFE_CALL( hipHostFree(temp_pos) );

      free(energy);
      free(v_temp);
    }

    CUDA_SAFE_CALL( hipMalloc((void**)&d_force,     inum*sizeof(float3)) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_energy,    inum*sizeof(float)) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_virial,    inum*3*sizeof(float3)) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_pos, nall*sizeof(float4)) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_type, nall*sizeof(int)) );
    
    init_neigh_list_gpu(d_neigh_list, inum, NEIGH_BIN_SIZE, szTailList);

    CUDA_SAFE_CALL( hipHostMalloc((void**)&temp_pos, nall*sizeof(float4)) );
    CUDA_SAFE_CALL( hipHostMalloc((void**)&f_temp,   inum*sizeof(float3)) );

    energy    = (float*) malloc(inum*sizeof(float));
    v_temp    = (float3*)malloc(inum*2*sizeof(float3));

    if (useCache) {
      bind_pos(d_pos, nall);
      bind_type(d_type, nall);
    }

    first_call = 0;
    CUDA_SAFE_CALL( hipDeviceSynchronize() );
    CUDA_SAFE_CALL( hipGetLastError() );
    CUDA_SAFE_CALL( hipMemcpy(d_type, host_type, nall*sizeof(int), 
			       hipMemcpyHostToDevice) );

  }

  TIMING( static double mallocTime = 0. );
  TIMING( mallocTime += cTimer2.GetET() );
  TIMING( printf("malloc time = %f ms\n", mallocTime*1e3) );

  TIMING( cTimer2.Start() );
  for (int i = 0; i < 3*nall; i+=3) { 
    temp_pos[i/3] = make_float4(atom_pos[i], atom_pos[i+1], atom_pos[i+2], 0.f);
  }

  TIMING( static double copyTime = 0. );
  TIMING( copyTime += cTimer2.GetET() );
  TIMING( printf("position copy time = %f ms\n", copyTime*1e3) );

  
  TIMING( cTimer2.Start() );
  CUDA_SAFE_CALL( hipMemcpy(d_pos, temp_pos, nall*sizeof(float4), hipMemcpyHostToDevice) );

  TIMING( static double h2dTime = 0. );
  TIMING( h2dTime += cTimer2.GetET() );
  TIMING( printf("h2d copy time = %f ms\n", h2dTime*1e3) );

  TIMING( cTimer2.Start() );
  if (ago == 0) {
    build_neigh_list_gpu(d_pos,
			 d_neigh_list,
			 h_boxlo, h_boxhi, cell_size,
			 inum, nall);
  }
  TIMING( static double neighTime = 0. );
  TIMING( neighTime += cTimer2.GetET() );
  TIMING( printf("Neigh List time = %f ms\n", neighTime*1e3) );

  TIMING( cTimer2.Start() );
  calc_lj_neigh_gpu(d_force, d_energy, d_virial,
		    d_pos, d_type,
		    d_neigh_list,
		    inum, nall,
		    eflag, vflag);
  TIMING( static double forceTime = 0. );
  TIMING( forceTime += cTimer2.GetET() );
  TIMING( printf("Force time = %f ms\n", forceTime*1e3) );
  TIMING( printf("GPU kernel time = %f ms\n", (forceTime + neighTime)*1e3) );


  TIMING( cTimer2.Start() );
  CUDA_SAFE_CALL( hipMemcpy(f_temp, d_force, inum*sizeof(float3), hipMemcpyDeviceToHost) );
  TIMING( static double d2hTime = 0. );
  TIMING( d2hTime += cTimer2.GetET() );
  TIMING( printf("d2h copy time = %f ms\n", d2hTime*1e3) );
  TIMING( printf("GPU-CPU data transfer time = %f ms\n", (h2dTime+d2hTime)*1e3) );

  TIMING( cTimer2.Start() );

  for (int i = 0; i < inum; i++) {
    force[i][0] += f_temp[i].x;
    force[i][1] += f_temp[i].y;
    force[i][2] += f_temp[i].z;
  }

  if (eflag) {
    CUDA_SAFE_CALL( hipMemcpy(energy, d_energy, 
			       inum*sizeof(float), hipMemcpyDeviceToHost) );
    for (int i = 0; i < inum; i++) {
      evdwl += energy[i];
    }
    evdwl *= 0.5f;
  }
  
  if (vflag) {
    CUDA_SAFE_CALL( hipMemcpy(v_temp, d_virial, inum*2*sizeof(float3), 
			       hipMemcpyDeviceToHost) ); 
    for (int i = 0; i < inum; i++) {
      virial[0] += v_temp[2*i].x;
      virial[1] += v_temp[2*i].y;
      virial[2] += v_temp[2*i].z;
      virial[3] += v_temp[2*i+1].x;
      virial[4] += v_temp[2*i+1].y;
      virial[5] += v_temp[2*i+1].z;
    }
    for (int i = 0; i < 6; i++) 
      virial[i] *= 0.5f;
  }


  TIMING( static double postTime = 0. );
  TIMING( postTime += cTimer2.GetET() );
  TIMING( printf("postprocess Time = %f ms\n", postTime*1e3) );
  TIMING( printf("Data process time = %f ms\n", (postTime+copyTime)*1e3) );

  TIMING( static double totalTime = 0. );
  TIMING( totalTime += cTimer.GetET() );
  TIMING( printf("lj_gpu time = %f ms\n", totalTime*1e3) );

  return evdwl;
 
}

EXTERN double lj_gpu_neigh(double **force, double *virial, 
			  double **host_x, int *host_type, 
			  const int inum, const int nall, 
			  const int ago, const bool eflag, const bool vflag, 
			  const double *boxlo, const double *boxhi) 
{
  return _lj_gpu_neigh<float,float>(force, virial, 
				    host_x, host_type, inum, nall, 
				    ago, eflag, vflag, boxlo, boxhi);
}


template <class numtyp, class acctyp>
double _lj_gpu_cell(double **force, double *virial,
		    double **host_x, int *host_type, const int inum, 
		    const int nall, const int ago, 
		    const bool eflag, const bool vflag, 
		    const double *boxlo, const double *boxhi)
{
  
  double evdwl=0.0;

  static int ncell = ncellx*ncelly*ncellz;

  static int first_call = 1;

  // allocate memory on CPU and GPU
  if (first_call || ago == 0) {
    if (!first_call) {
     if (useCache) {
	unbind_pos();
	unbind_type();
      }

      free(energy);
      free(v_temp);
      
      CUDA_SAFE_CALL( hipFree(d_force) );
      CUDA_SAFE_CALL( hipFree(d_energy) );
      CUDA_SAFE_CALL( hipFree(d_virial) );

      CUDA_SAFE_CALL( hipFree(d_pos) );
      CUDA_SAFE_CALL( hipFree(d_type) );
      CUDA_SAFE_CALL( hipHostFree(f_temp) );
      CUDA_SAFE_CALL( hipHostFree(temp_pos) );

      clear_cell_list_gpu(d_cell_list);
    }

    energy    = (float*) malloc(inum*sizeof(float));
    v_temp    = (float3*)malloc(inum*2*sizeof(float3));


    hipMalloc((void**)&d_force,     inum*sizeof(float3));
    hipMalloc((void**)&d_energy,    inum*sizeof(float));
    hipMalloc((void**)&d_virial,    inum*3*sizeof(float3));

    CUDA_SAFE_CALL( hipMalloc((void**)&d_pos, nall*sizeof(float4)) );
    CUDA_SAFE_CALL( hipMalloc((void**)&d_type, nall*sizeof(int)) );

    CUDA_SAFE_CALL( hipHostMalloc((void**)&f_temp,   inum*sizeof(float3)) );
    CUDA_SAFE_CALL( hipHostMalloc((void**)&temp_pos, nall*sizeof(float4)) );

    init_cell_list_gpu(d_cell_list, nall, ncell);

    CUDA_SAFE_CALL( hipMemcpy(d_type, host_type, nall*sizeof(int), 
			       hipMemcpyHostToDevice) );

    if (useCache) {
      bind_pos(d_pos, nall);
      bind_type(d_type, nall);
    }

    first_call = 0;
  }

  /* build cell-list on GPU */
  double *atom_pos = host_x[0];
  for (int i = 0; i < 3*nall; i+=3) { 
    temp_pos[i/3] = make_float4(atom_pos[i], atom_pos[i+1], atom_pos[i+2], 0.f);
  }
  CUDA_SAFE_CALL( hipMemcpy(d_pos, temp_pos, nall*sizeof(float4), 
			     hipMemcpyHostToDevice) );
  if (ago == 0) {
    build_cell_list_gpu(d_pos, d_cell_list, h_boxlo, h_boxhi, 
			cell_size, inum, nall);
  }

  calc_lj_cell_gpu(d_force, d_energy, d_virial,
		   d_pos, d_type, d_cell_list,
		   inum, nall, ncellx, 
		   ncelly, ncellz, cell_size,
		   eflag, vflag);

  CUDA_SAFE_CALL( hipMemcpy(f_temp, d_force, inum*sizeof(float3), 
			     hipMemcpyDeviceToHost) );

  for (int i = 0; i < inum; i++) {
    force[i][0] += f_temp[i].x;
    force[i][1] += f_temp[i].y;
    force[i][2] += f_temp[i].z;
  }
  
  if (eflag) {
    CUDA_SAFE_CALL( hipMemcpy(energy, d_energy, 
			       inum*sizeof(float), hipMemcpyDeviceToHost) );
    for (int i = 0; i < inum; i++) {
      evdwl += energy[i];
    }
    evdwl *= 0.5f;
  }
  
  if (vflag) {
    CUDA_SAFE_CALL( hipMemcpy(v_temp, d_virial, inum*2*sizeof(float3), 
			       hipMemcpyDeviceToHost) ); 
    for (int i = 0; i < inum; i++) {
      virial[0] += v_temp[2*i].x;
      virial[1] += v_temp[2*i].y;
      virial[2] += v_temp[2*i].z;
      virial[3] += v_temp[2*i+1].x;
      virial[4] += v_temp[2*i+1].y;
      virial[5] += v_temp[2*i+1].z;
    }
    for (int i = 0; i < 6; i++) 
      virial[i] *= 0.5f;
  }

  return evdwl; 
}

EXTERN double lj_gpu_cell(double **force, double *virial, 
			  double **host_x, int *host_type, 
			  const int inum, const int nall, 
			  const int ago, const bool eflag, const bool vflag, 
			  const double *boxlo, const double *boxhi) 
{
  return _lj_gpu_cell<float,float>(force, virial, 
				   host_x, host_type, inum, nall, 
				   ago, eflag, vflag, boxlo, boxhi);
}

EXTERN void lj_gpu_time() {
  /*  cout.precision(4);
  cout << "Atom copy:     " << LJMF.time_atom.total_seconds() << " s.\n";
  cout << "Neighbor copy: " << LJMF.time_nbor.total_seconds() << " s.\n";
  cout << "LJ calc:       " << LJMF.time_pair.total_seconds() << " s.\n";*/
  //cout << "Answer copy:   " << LJMF.time_answer.total_seconds() << " s.\n";
}

EXTERN int lj_gpu_num_devices() {
  int num_devices;
  CUDA_SAFE_CALL( hipGetDeviceCount(&num_devices) );
  return num_devices;
}

EXTERN double lj_gpu_bytes() {
  return 0.0;
}
