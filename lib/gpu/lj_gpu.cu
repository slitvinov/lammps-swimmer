#include "hip/hip_runtime.h"
/* ----------------------------------------------------------------------
   LAMMPS - Large-scale Atomic/Molecular Massively Parallel Simulator
   http://lammps.sandia.gov, Sandia National Laboratories
   Steve Plimpton, sjplimp@sandia.gov

   Copyright (2003) Sandia Corporation.  Under the terms of Contract
   DE-AC04-94AL85000 with Sandia Corporation, the U.S. Government retains
   certain rights in this software.  This software is distributed under 
   the GNU General Public License.

   See the README file in the top-level LAMMPS directory.
------------------------------------------------------------------------- */

/* ----------------------------------------------------------------------
   Contributing authors: Mike Brown (SNL), wmbrown@sandia.gov
                         Peng Wang (Nvidia), penwang@nvidia.com
                         Paul Crozier (SNL), pscrozi@sandia.gov
------------------------------------------------------------------------- */

#include <iostream>
#include <cassert>
#include "nvc_macros.h"
#include "nvc_timer.h"
#include "nvc_device.h"
#include "pair_gpu_texture.h"
#include "pair_gpu_cell.h"
#include "lj_gpu_memory.cu"
#include "lj_gpu_kernel.h"

#ifdef WINDLL
#include <windows.h>
BOOL APIENTRY DllMain(HANDLE hModule, DWORD dwReason, LPVOID lpReserved)
{
    return TRUE;
}
#endif

#ifdef WINDLL
#define EXTERN extern "C" __declspec(dllexport) 
#else
#define EXTERN 
#endif
using namespace std;

static LJ_GPU_Memory<PRECISION,ACC_PRECISION> LJMF;
#define LJMT LJ_GPU_Memory<numtyp,acctyp>

static float kernelTime = 0.0;
static int ncell1D;
static float *energy, *d_energy;
static float3 *d_force, *f_temp, *v_temp, *d_virial;
static cell_list cell_list_gpu;

// ---------------------------------------------------------------------------
// Convert something to a string
// ---------------------------------------------------------------------------
#include <sstream>

template <class t>
inline string lj_gpu_toa(const t& in) {
  ostringstream o;
  o.precision(2);
  o << in;
  return o.str();
}

// ---------------------------------------------------------------------------
// Return string with GPU info
// ---------------------------------------------------------------------------
EXTERN void lj_gpu_name(const int id, const int max_nbors, char * name) {
  string sname=LJMF.gpu.name(id)+", "+
              lj_gpu_toa(LJMF.gpu.cores(id))+" cores, "+
              lj_gpu_toa(LJMF.gpu.gigabytes(id))+" GB, "+
              lj_gpu_toa(LJMF.gpu.clock_rate(id))+" GHZ";
  strcpy(name,sname.c_str());
}

// ---------------------------------------------------------------------------
// Allocate memory on host and device and copy constants to device
// ---------------------------------------------------------------------------
EXTERN bool lj_gpu_init(int &ij_size, const int ntypes, double **cutsq,double **sigma, 
			 double **epsilon, double **host_lj1, double **host_lj2, 
			 double **host_lj3, double **host_lj4, double **offset, 
			 double *special_lj, double *boxlo, double *boxhi, 
			 double cell_size, double skin,
			 const int max_nbors, const int gpu_id) {
  LJMF.gpu.init();
  if (LJMF.gpu.num_devices()==0)
    return false;                   

  ij_size=IJ_SIZE;

  bool ret = LJMF.init(ij_size, ntypes, cutsq, sigma, epsilon, host_lj1, host_lj2, 
		       host_lj3, host_lj4, offset, special_lj, max_nbors, gpu_id);

  ncell1D = ceil(((boxhi[0] - boxlo[0]) + 2.0*cell_size) / cell_size);

  init_cell_list_const(cell_size, skin, boxlo, boxhi);

  return ret;
}

// ---------------------------------------------------------------------------
// Clear memory on host and device
// ---------------------------------------------------------------------------
EXTERN void lj_gpu_clear() {
  free(energy);
  free(v_temp);
  hipHostFree(f_temp);
  hipFree(d_force);
  hipFree(d_energy);
  hipFree(d_virial);
  clear_cell_list(cell_list_gpu);

  LJMF.clear();
}


// ---------------------------------------------------------------------------
// Calculate energies and forces for all ij interactions
// ---------------------------------------------------------------------------
template <class numtyp, class acctyp>
void _lj_gpu(LJMT &ljm, const bool eflag, const bool vflag, const bool rebuild){
  // Compute the block size and grid size to keep all cores busy
  const int BX=BLOCK_1D;

  int GX=static_cast<int>(ceil(static_cast<double>(ljm.atom.inum())/BX));

  ljm.time_pair.start();

  if (ljm.shared_types)
    kernel_lj_fast<numtyp,acctyp><<<GX,BX,0,ljm.pair_stream>>>
           (ljm.special_lj.begin(), ljm.nbor.dev_nbor.begin(), 
            ljm.nbor.ij.begin(), ljm.nbor.dev_nbor.row_size(), 
            ljm.atom.ans.begin(), ljm.atom.ans.row_size(), eflag,
            vflag, ljm.atom.inum(), ljm.atom.nall());
  else
    kernel_lj<numtyp,acctyp><<<GX,BX,0,ljm.pair_stream>>>
           (ljm.special_lj.begin(), ljm.nbor.dev_nbor.begin(), 
            ljm.nbor.ij.begin(), ljm.nbor.dev_nbor.row_size(), 
            ljm.atom.ans.begin(), ljm.atom.ans.row_size(), eflag, 
            vflag, ljm.atom.inum(), ljm.atom.nall());
	    ljm.time_pair.stop();
}

EXTERN void lj_gpu(const bool eflag, const bool vflag, const bool rebuild) {
  _lj_gpu<PRECISION,ACC_PRECISION>(LJMF, eflag,vflag,rebuild);
}

template <class numtyp, class acctyp>
double _lj_gpu_cell(LJMT &ljm, double **force, double *virial,
		    double **host_x, int *host_type, const int inum, 
		    const int nall, const int ago, const bool eflag, const bool vflag, 
		    const double *boxlo, const double *boxhi)
{
  ljm.atom.nall(nall);
  ljm.atom.inum(inum);

  ljm.nbor.time_nbor.start();
  ljm.nbor.time_nbor.stop();

  double evdwl=0.0;

  static int buffer = CELL_SIZE;
  static int ncell = (int)pow((float)ncell1D,3);

  static int first_call = 1;

  // allocate memory on CPU and GPU
  if (first_call) {
    energy    = (float*) malloc(inum*sizeof(float));
    v_temp    = (float3*)malloc(inum*2*sizeof(float3));
    hipHostMalloc((void**)&f_temp,   inum*sizeof(float3));

    hipMalloc((void**)&d_force,     inum*sizeof(float3));
    hipMalloc((void**)&d_energy,    inum*sizeof(float));
    hipMalloc((void**)&d_virial,    inum*3*sizeof(float3));

    init_cell_list(cell_list_gpu, nall, ncell, buffer);

    first_call = 0;
  }

  if (!first_call && ago == 0) {
    free(energy);
    free(v_temp);
    hipHostFree(f_temp);
    hipFree(d_force);
    hipFree(d_energy);
    hipFree(d_virial);

    energy    = (float*) malloc(inum*sizeof(float));
    v_temp    = (float3*)malloc(inum*2*sizeof(float3));
    hipHostMalloc((void**)&f_temp,   inum*sizeof(float3));

    hipMalloc((void**)&d_force,     inum*sizeof(float3));
    hipMalloc((void**)&d_energy,    inum*sizeof(float));
    hipMalloc((void**)&d_virial,    inum*3*sizeof(float3));

    clear_cell_list(cell_list_gpu);
    init_cell_list(cell_list_gpu, nall, ncell, buffer);
  }

  // build cell-list on GPU
  ljm.atom.time_atom.start();
  build_cell_list(host_x[0], host_type, cell_list_gpu, 
		  ncell, ncell1D, buffer, inum, nall, ago);
  ljm.atom.time_atom.stop();

  ljm.time_pair.start();

#ifdef TIMING
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
#endif

  // call the cell-list force kernel
  const int BX=BLOCK_1D;
  dim3 GX(ncell1D, ncell1D*ncell1D);
  if (eflag == 0 && vflag == 0) {
    kernel_lj_cell<false,false><<<GX, BX, 0>>>
      (d_force, d_energy, d_virial, 
       cell_list_gpu.pos, 
       cell_list_gpu.idx, 
       cell_list_gpu.type, 
       cell_list_gpu.natom,
       inum, nall, ncell);
  } else {
    kernel_lj_cell<true,true><<<GX, BX, 3*sizeof(float)*MAX_SHARED_TYPES*MAX_SHARED_TYPES>>>
      (d_force, d_energy, d_virial, 
       cell_list_gpu.pos, 
       cell_list_gpu.idx, 
       cell_list_gpu.type, 
       cell_list_gpu.natom,
       inum, nall, ncell);
  }

#ifdef TIMING
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float kTime;
  hipEventElapsedTime(&kTime, start, stop);
  kernelTime += kTime;
  printf("kernelTime = %f, eflag=%d, vflag=%d\n", kTime, eflag, vflag);
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif

  // copy results from GPU to CPU
  hipMemcpy(f_temp, d_force, inum*sizeof(float3), hipMemcpyDeviceToHost);
  if (eflag) {
    hipMemcpy(energy, d_energy, inum*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < inum; i++) {
      evdwl += energy[i];
    }
    evdwl *= 0.5f;
  }
  if (vflag) {
    hipMemcpy(v_temp, d_virial, inum*2*sizeof(float3), hipMemcpyDeviceToHost);
    for (int i = 0; i < inum; i++) {
      virial[0] += v_temp[2*i].x;
      virial[1] += v_temp[2*i].y;
      virial[2] += v_temp[2*i].z;
      virial[3] += v_temp[2*i+1].x;
      virial[4] += v_temp[2*i+1].y;
      virial[5] += v_temp[2*i+1].z;
    }
    for (int i = 0; i < 6; i++) 
      virial[i] *= 0.5f;
  }

  for (int i = 0; i < inum; i++) {
    force[i][0] += f_temp[i].x;
    force[i][1] += f_temp[i].y;
    force[i][2] += f_temp[i].z;
  }

  ljm.time_pair.stop();

  ljm.atom.time_atom.add_to_total();
  ljm.nbor.time_nbor.add_to_total();
  ljm.time_pair.add_to_total();


  return evdwl;
 
}

EXTERN double lj_gpu_cell(double **force, double *virial, double **host_x, int *host_type, const int inum, const int nall, 
		   const int ago, const bool eflag, const bool vflag, 
		   const double *boxlo, const double *boxhi) 
{
  return _lj_gpu_cell<PRECISION,ACC_PRECISION>(LJMF, force, virial, host_x, host_type, inum, nall, 
					       ago, eflag, vflag, boxlo, boxhi);
}

template <class numtyp, class acctyp>
double _lj_gpu_n2(LJMT &ljm, double **force, double *virial,
		  double **host_x, int *host_type, const int inum, const int nall, const bool eflag, const bool vflag,
		  const double *boxlo, const double *boxhi)
{
  ljm.atom.nall(nall);
  ljm.atom.inum(inum);


  ljm.nbor.time_nbor.start();
  ljm.nbor.time_nbor.stop();

  
  double evdwl=0.0;

#ifdef NOUSE
  static int first_call = 1;

  if (first_call) {
    energy    = (float*)    malloc(inum*sizeof(float));
    v_temp    = (float3*)   malloc(inum*2*sizeof(float3));
    hipHostMalloc((void**)&f_temp,   inum*sizeof(float3));
    hipHostMalloc((void**)&pos_temp, nall*sizeof(float3));
    hipMalloc((void**)&d_force,     inum*sizeof(float3));
    hipMalloc((void**)&d_energy,    inum*sizeof(float));
    hipMalloc((void**)&d_virial,    inum*3*sizeof(float3));
    hipMalloc((void**)&d_pos,       nall*sizeof(float3));
    hipMalloc((void**)&d_type,      nall*sizeof(int));
    first_call = 0;
  }


  ljm.atom.time_atom.start();
  double *atom_pos = host_x[0];
  for (int i = 0; i < 3*nall; i+=3) { 
    pos_temp[i/3] = make_float3(atom_pos[i], atom_pos[i+1], atom_pos[i+2]);
  }
  hipMemcpy(d_pos, pos_temp, nall*sizeof(float3), hipMemcpyHostToDevice);
  hipMemcpy(d_type, host_type, nall*sizeof(int),  hipMemcpyHostToDevice);

  ljm.atom.time_atom.stop();

  ljm.time_pair.start();
  
  // Compute the block size and grid size to keep all cores busy
  const int BX=BLOCK_1D;
  dim3 GX(static_cast<int>(ceil(static_cast<double>(inum)/BX)));

#ifdef TIMING
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipEventRecord(start, 0);
#endif

  // N^2 force kernel
  kernel_lj_n2<numtyp, acctyp><<<GX, BX>>>(d_force, d_energy, d_virial, 
					   d_pos, d_type, eflag, vflag, inum, nall);

#ifdef TIMING
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  float kTime;
  hipEventElapsedTime(&kTime, start, stop);
  kernelTime += kTime;
  printf("kernelTime = %f, eflag=%d, vflag=%d\n", kTime, eflag, vflag);
  hipEventDestroy(start);
  hipEventDestroy(stop);
#endif

  // copy results from GPU to CPU
  hipMemcpy(f_temp, d_force, inum*sizeof(float3), hipMemcpyDeviceToHost);
  if (eflag) {
    hipMemcpy(energy, d_energy, inum*sizeof(float), hipMemcpyDeviceToHost);
    for (int i = 0; i < inum; i++) {
      evdwl += energy[i];
    }
    evdwl *= 0.5f;
  }
  if (vflag) {
    hipMemcpy(v_temp, d_virial, inum*2*sizeof(float3), hipMemcpyDeviceToHost);
    for (int i = 0; i < inum; i++) {
      virial[0] += v_temp[2*i].x;
      virial[1] += v_temp[2*i].y;
      virial[2] += v_temp[2*i].z;
      virial[3] += v_temp[2*i+1].x;
      virial[4] += v_temp[2*i+1].y;
      virial[5] += v_temp[2*i+1].z;
    }
    for (int i = 0; i < 6; i++) 
      virial[i] *= 0.5f;
  }

  for (int i = 0; i < inum; i++) {
    force[i][0] += f_temp[i].x;
    force[i][1] += f_temp[i].y;
    force[i][2] += f_temp[i].z;
  }
#endif
  ljm.time_pair.stop();

  ljm.atom.time_atom.add_to_total();
  ljm.nbor.time_nbor.add_to_total();
  ljm.time_pair.add_to_total();

  return evdwl;
}

EXTERN double lj_gpu_n2(double **force, double *virial, double **host_x, int *host_type, const int inum, const int nall, 
		 const bool eflag, const bool vflag,
		 const double *boxlo, const double *boxhi) 
{
  return _lj_gpu_n2<PRECISION,ACC_PRECISION>(LJMF, force, virial, host_x, host_type, inum, nall, 
					  eflag, vflag, boxlo, boxhi);
}

EXTERN void lj_gpu_time() {
  cout.precision(4);
  cout << "Atom copy:     " << LJMF.atom.time_atom.total_seconds() << " s.\n";
  cout << "Neighbor copy: " << LJMF.nbor.time_nbor.total_seconds() << " s.\n";
  cout << "LJ calc:       " << LJMF.time_pair.total_seconds() << " s.\n";
  cout << "Answer copy:   " << LJMF.atom.time_answer.total_seconds() << " s.\n";
}

EXTERN int lj_gpu_num_devices() {
  return LJMF.gpu.num_devices();
}

EXTERN double lj_gpu_bytes() {
  return LJMF.host_memory_usage();
}
